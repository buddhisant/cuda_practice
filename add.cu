/*利用cuda完成两个1024*1024矩阵的加法*/
#include<iostream>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#include"hip/hip_runtime.h"

#define cols 1024
#define rows 1024

using namespace std;

__global__ void Add(float** Ad,float** Bd,float** Cd)
{
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	if(x<cols && y<rows)
	{
		Cd[x][y]=Ad[x][y]+Bd[x][y];
	}
}

int main()
{
	struct timeval start, end;
	gettimeofday( &start, NULL);
	float **A,**B,**C,**Ad,**Bd,**Cd;
	float *a,*b,*c,*ad,*bd,*cd;
	int n=rows * cols;

	A=new float* [cols];
	B=new float* [cols];
	C=new float* [cols];
	a=new float [n];
	b=new float [n];
	c=new float [n];

	hipMalloc((void**)&Ad,sizeof(float*)*cols);
	hipMalloc((void**)&Bd,sizeof(float*)*cols);
	hipMalloc((void**)&Cd,sizeof(float*)*cols);
	hipMalloc((void**)&ad,sizeof(float)*n);
	hipMalloc((void**)&bd,sizeof(float)*n);
	hipMalloc((void**)&cd,sizeof(float)*n);

	for(int i=0;i<n;i++)
	{
		a[i]=90.0;
		b[i]=10.0;
	}
	for(int i=0;i<cols;i++)
	{
		//ad, bd, cd是一维向量，如果在gpu上按照二维矩阵进行运算，则需要将其和Ad, Bd, Cd建立对应关系，建立对应关系的过程在cpu上完成
		A[i]=ad+i*rows;
		B[i]=bd+i*rows;
		C[i]=cd+i*rows;
	}

	hipMemcpy(Ad,A,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(Cd,C,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(ad,a,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(bd,b,n*sizeof(float),hipMemcpyHostToDevice);

	dim3 dimBlock(16,16);
	dim3 dimGrid(cols/16+1,rows/16+1);
	Add<<<dimGrid,dimBlock>>>(Ad,Bd,Cd);

	hipMemcpy(c,cd,n*sizeof(float),hipMemcpyDeviceToHost);

	float target=100.0;
	float error=0.0;
	for(int i=0;i<n;i++)
	{
		error+=abs(target-c[i]);
	}
	cout<<"total error is "<<error<<endl;
	delete [] a;
	delete [] b;
	delete [] c;
	delete [] A;
	delete [] B;
	delete [] C;
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	gettimeofday( &end, NULL );
	int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
	cout << "total time is " << timeuse/1000 << "ms" <<endl;
	return 0;
}
