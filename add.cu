//#include <torch/torch.h>
//#include <iostream>
//
//#include <ATen/ATen.h>
//#include <ATen/cuda/CUDAContext.h>
//
//#include <THC/THC.h>
//#include <THC/THCDeviceUtils.cuh>
//
//#include <vector>
//#include <iostream>
//#include <cmath>
//
//#include <torch/extension.h>
//
//#define CHECK_CUDA(x) \
//	TORCH_CHECK(x.device().is_cuda(), #x "must be a CUDA tensor")
//
//int main() {
//  at::Tensor tensor = at::rand({2, 3});
//  at::Tensor tensor_gpu = tensor.cuda();
//  CHECK_CUDA(tensor_gpu);
//  std::cout << tensor_gpu << std::endl;
//  return 0;
//}

#include<iostream>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#include"hip/hip_runtime.h"

#define cols 1024
#define rows 1024

using namespace std;

__global__ void Add(float** Ad,float** Bd,float** Cd)
{
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	if(x<cols && y<rows)
	{
		Cd[x][y]=Ad[x][y]+Bd[x][y];
	}
}

int main()
{
	struct timeval start, end;
	gettimeofday( &start, NULL);
	float **A,**B,**C,**Ad,**Bd,**Cd;
	float *a,*b,*c,*ad,*bd,*cd;
	int n=rows * cols;

	A=new float* [cols];
	B=new float* [cols];
	C=new float* [cols];
	a=new float [n];
	b=new float [n];
	c=new float [n];

	hipMalloc((void**)&Ad,sizeof(float*)*cols);
	hipMalloc((void**)&Bd,sizeof(float*)*cols);
	hipMalloc((void**)&Cd,sizeof(float*)*cols);
	hipMalloc((void**)&ad,sizeof(float)*n);
	hipMalloc((void**)&bd,sizeof(float)*n);
	hipMalloc((void**)&cd,sizeof(float)*n);

	for(int i=0;i<n;i++)
	{
		a[i]=90.0;
		b[i]=10.0;
	}
	for(int i=0;i<cols;i++)
	{
		A[i]=ad+i*rows;
		B[i]=bd+i*rows;
		C[i]=cd+i*rows;
	}

	hipMemcpy(Ad,A,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(Cd,C,cols*sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(ad,a,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(bd,b,n*sizeof(float),hipMemcpyHostToDevice);

	dim3 dimBlock(16,16);
	dim3 dimGrid(cols/16+1,rows/16+1);
	Add<<<dimGrid,dimBlock>>>(Ad,Bd,Cd);

	hipMemcpy(c,cd,n*sizeof(float),hipMemcpyDeviceToHost);

	float target=100.0;
	float error=0.0;
	for(int i=0;i<n;i++)
	{
		error+=abs(target-c[i]);
	}
	cout<<"total error is "<<error<<endl;
	delete [] a;
	delete [] b;
	delete [] c;
	delete [] A;
	delete [] B;
	delete [] C;
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	gettimeofday( &end, NULL );
	int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
	cout << "total time is " << timeuse/1000 << "ms" <<endl;
	return 0;
}
