/*用gpu实现2个矩阵之间的乘法*/
#include<iostream>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#include"hip/hip_runtime.h"

using namespace std;

#define cols 1024
#define rows 1024

__global__ void multiply(float**Ad,float**Bd,float**Cd)
{
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	if(x<rows && y<cols)
	{
		for(int i=0;i<cols;i++)
		{
				Cd[y][x]+=Ad[y][i]*Bd[i][x];
		}
	}
}

int main()
{
	struct timeval start, end;
	int n=cols*rows;
	float **A,**B,**C,**Ad,**Bd,**Cd;
	float *a,*b,*c,*ad,*bd,*cd;
	A=new float* [cols];
	B=new float* [cols];
	C=new float* [cols];
	a=new float [n];
	b=new float [n];
	c=new float [n];

	hipMalloc((void**)&Ad,sizeof(float*)*cols);
	hipMalloc((void**)&Bd,sizeof(float*)*cols);
	hipMalloc((void**)&Cd,sizeof(float*)*cols);
	hipMalloc((void**)&ad,sizeof(float)*n);
	hipMalloc((void**)&bd,sizeof(float)*n);
	hipMalloc((void**)&cd,sizeof(float)*n);

	for(int i=0;i<n;i++)
	{
		a[i]=2;
		b[i]=2;
	}

	for(int i=0;i<cols;i++)
	{
		A[i]=ad+i*rows;
		B[i]=bd+i*rows;
		C[i]=cd+i*rows;
	}

	gettimeofday( &start, NULL);//以开始向gpu拷贝数据为起点，记录时间
	hipMemcpy(Ad,A,sizeof(float*)*cols,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeof(float*)*cols,hipMemcpyHostToDevice);
	hipMemcpy(Cd,C,sizeof(float*)*cols,hipMemcpyHostToDevice);
	hipMemcpy(ad,a,sizeof(float)*n,hipMemcpyHostToDevice);
	hipMemcpy(bd,b,sizeof(float)*n,hipMemcpyHostToDevice);

	dim3 dimBlock(16,16);
	dim3 dimGrid(cols/16+1,rows/16+1);
	multiply<<<dimGrid,dimBlock>>>(Ad,Bd,Cd);
	hipMemcpy(c,cd,sizeof(float)*n,hipMemcpyDeviceToHost);
	gettimeofday( &end, NULL );//以从gpu返回计算数据为终点，记录时间

	float target=4096;
	float error=0.0;
	for(int i=0;i<n;i++)
	{
		error+=abs(c[i]-target);
	}
	cout<<"error is "<<error<<endl;

	int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
	cout << "total time is " << timeuse/1000 << "ms" <<endl;
	delete [] a;
	delete [] b;
	delete [] c;
	delete [] A;
	delete [] B;
	delete [] C;

	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	return 0;
}
