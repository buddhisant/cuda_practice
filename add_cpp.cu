/*用cpu实现2个矩阵之间的加法*/
#include<iostream>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#include"hip/hip_runtime.h"

using namespace std;

#define cols 1024
#define rows 1024

int main()
{
	struct timeval start, end;
	int n=cols*rows;
	float **A,**B,**C;
	float *a,*b,*c;
	A=new float* [cols];
	B=new float* [cols];
	C=new float* [cols];
	a=new float [n];
	b=new float [n];
	c=new float [n];

	for(int i=0;i<n;i++)
	{
		a[i]=2;
		b[i]=2;
	}

	for(int i=0;i<cols;i++)
	{
		A[i]=a+i*rows;
		B[i]=b+i*rows;
		C[i]=c+i*rows;
	}

	gettimeofday( &start, NULL);
	for(int i=0;i<rows;i++)
	{
		for(int j=0;j<cols;j++)
		{
			C[i][j]+=A[i][j]+B[i][j];
		}
	}
	gettimeofday( &end, NULL );

	float target=4.0;
	float error=0.0;
	for(int i=0;i<rows;i++)
	{
		for(int j=0;j<cols;j++)
		{
			error+=abs(C[i][j]-target);
		}
	}
	cout<<"error is "<<error<<endl;

	int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
	cout << "total time is " << timeuse/1000 << "ms" <<endl;
	delete [] a;
	delete [] b;
	delete [] c;
	delete [] A;
	delete [] B;
	delete [] C;

	return 0;
}
